#include "hip/hip_runtime.h"
//Set to 0 for added slow serial execution, 1 for only fast execution (debugging)
#define DEBUG 0

#define iterations		1000
#define N_mat			13 // mat size
#define MAX_acc			14 // extra thread for addition of non matrix mults
#define threads_per_block 4

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

//#include <device_atomic_functions.hpp>

__global__ void matrixMulCUDASlow(int *A, int *B, int *C) {

	for (uint8_t i = 0; i < N_mat; i++) {
		// ROW OPERATIONS
		for (uint8_t j = 0; j < N_mat; j++) {
			// COLUMN OPERATIONS
			int32_t Sum = 0;
			// CALCULATE DOT PRODUCT
			for (uint8_t k = 0; k < N_mat; k++) {
				Sum += A[(i * N_mat) + k] * B[(k * N_mat) + j];
			}
			C[(i * N_mat) + j] = Sum + A[(i * N_mat) + j] + B[(i * N_mat) + j];
		}
	}
}
__global__ void matrixMulCUDA(int *A, int *B, int *C)
{
	// Thread index
	int row = threadIdx.x + threads_per_block*blockIdx.x; // offset row with block
	int col = threadIdx.y;
	int	z = threadIdx.z;
	int index = (row * N_mat) + col;
	int offset = 2 * z;

	int multi[N_mat*N_mat];
	if (z == 0) multi[index] = 0;
	__syncthreads();

	if (row < 13) {// prevent rows larger than 12 to write, as these don't exist
		if (z == MAX_acc) { // extra thread
			atomicAdd(&multi[index], A[index] + B[index]);
		}
		else {
			atomicAdd(&multi[index], A[(row * N_mat) + offset] * B[col + (N_mat * offset)]);
		}
	}
	__syncthreads();
	if (z == MAX_acc && row < 13) { // let one thread push
		atomicExch(&C[index], multi[index]);
	}
}

void printmatrix(int m[N_mat*N_mat]) {
	for (size_t i = 0; i < N_mat; i++) {
		for (size_t j = 0; j < N_mat; j++) {
			std::cout << m[(N_mat * i) + j];
			std::cout << ",";
		}
		std::cout << "\n";
	}
}


int main()
{
	int a[N_mat*N_mat] = {
		28, 122,  80,   42,   54,   122,  98,   42,   99,   58,   124,  29,   21 ,
		113,  85,   30,   35,   41,   98,   103,  68,   15,   50,   31,   80,   54,
		47, 37,   23,   96,   59,   47,   84,   26,   84,   72,   51,   118,  119 ,
		38, 121,  45,   21,   87,   91,   20,   69,   98,   119,  15,   89,   47 ,
		40, 71,   105,  76,   31,   65,   109,  30,   127,  110,  17,   64,   64 ,
		45, 20,   113,  86,   86,   51,   104,  115,  61,   103,  60,   113,  44 ,
		101,  107,  33,   63,   39,   47,   120,  20,   41,   64,   102,  59,   86 ,
		9,  42,   118,  26,   83,   123,  10,   82,   47,   108,  127,  4,    66 ,
		75, 26,   117,  80,   47,   111,  38,   22,   98,   101,  92,   100,  48 ,
		90, 9,    71,   36,   90,   95,   4,    94,   72,   29,   77,   118,  78 ,
		81, 75,   97,   127,  22,   8,    96,   80,   100,  88,   69,   114,  16 ,
		25, 109,  74,   3,    126,  56,   99,   15,   69,   73,   76,   19,   97 ,
		59, 84,   102,  53,   30,   34,   33,   105,  75,   102,  60,   121,  93
	};

	int b[N_mat*N_mat] = {
		102,  61,   111,  79,   99,   3,    25,   50,   33,   48,   5,    94,   28 ,
		106,  89,   35,   37,   112   ,51   ,13,  70,   3,    110,  31,   7,    99 ,
		65, 115,  94,   68,   95,   114,  34,   34,   64,   1,    11,   66,   126 ,
		114,  37,   42,   3,    88,   35,   124   ,50,  74,   95,   25,   34    ,24 ,
		25, 111,  4,    116   ,54,  90    ,11   ,32   ,121, 20,   26,   62,   60 ,
		45, 41,   20,   33,   89,   75,   89,   2,    28    ,19,  96,   46,   119 ,
		39, 68,   87,   59,   33,   82    ,94,  14,   115,  0,    0,    92,   85 ,
		58, 62,   122,  106   ,93,  39,   86,   80,   75,   23,   57,   89,   7 ,
		119,  75,   20,   42,   1,    120,  83,   24,   62,   78,   20,   25,   126 ,
		121,  42,   78,   45,   8,    17,   52,   38,   44,   13,   104,  57,   62 ,
		29, 96,   0,    64,   47,   50,   22,   17,   88,   63,   108,  78,   101 ,
		70, 108,  69,   12,   0,    80,   115   ,107, 71,   54,   5,    57,   3 ,
		123,  72,   56,   5,    30,   45    ,2,   11,   124,  84,   63,   47,   104
	};

	int c[N_mat*N_mat] = { 0 };
	int cslow[N_mat*N_mat] = { 0 };

	int *dev_a, *dev_b, *dev_c, *dev_c_slow;


	//Initialize Timer
	hipEvent_t start, start1, stop, stop1;
	hipEventCreate(&start);
	hipEventCreate(&start1);
	hipEventCreate(&stop);
	hipEventCreate(&stop1);

	//Device Info
	hipDeviceProp_t prop;
	int device;
	hipGetDevice(&device);
	hipGetDeviceProperties(&prop, device);

	printf("  Device name: %s\n", prop.name);
	printf("  clockRate: %i\n", prop.clockRate);
	printf("  warpSize: %i\n", prop.warpSize);
	printf("  multiProcessorCount: %i\n", prop.multiProcessorCount);
	printf("  maxThreadsDim: %ix%ix%i\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("  maxThreadsPerBlock: %i\n", prop.maxThreadsPerBlock);
	printf("  maxThreadsPerMultiProcessor: %i\n", prop.maxThreadsPerMultiProcessor);
	std::cout << std::endl;




	//Allocating vectors in device memory
	hipMalloc((void**)&dev_a, N_mat*N_mat * sizeof(int));
	hipMalloc((void**)&dev_b, N_mat*N_mat * sizeof(int));
	hipMalloc((void**)&dev_c, N_mat*N_mat * sizeof(int));
	hipMalloc((void**)&dev_c_slow, N_mat*N_mat * sizeof(int));


	//Copy vectors from host memory to device memory
	hipMemcpy(dev_a, a, N_mat*N_mat * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N_mat*N_mat * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, N_mat*N_mat * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c_slow, c, N_mat*N_mat * sizeof(int), hipMemcpyHostToDevice);


	//Invoce kernel
	// Setup execution parameters
	dim3 threads(3, N_mat, MAX_acc);
	//dim3 grid(13 / threads.x, 13 / threads.y);

	//Fast Parallel Execution
	hipEventRecord(start);
	//matrixMulCUDA <<<1,threads>>> (dev_a, dev_b, dev_c);
	matrixMulCUDA << <iterations, threads >> > (dev_a, dev_b, dev_c);
	hipDeviceSynchronize();
	hipEventRecord(stop);

	//Occupancy
	int block_size = N_mat*N_mat;
	int output = 1;
	hipOccupancyMaxActiveBlocksPerMultiprocessor(
		&output,
		matrixMulCUDA,
		block_size,
		0);

	double activeWarps = (double)output * (double)block_size / (double)prop.warpSize;
	double maxWarps = (double)prop.maxThreadsPerMultiProcessor / (double)prop.warpSize;

	std::cout << "Occupancy (fast): " << (double)activeWarps / maxWarps * 100 << "%" << std::endl;
	std::cout << std::endl;

	//Fast print
	std::cout << "Fast Parallel Execution:\n";
	unsigned long mem_size_C = sizeof(int) * N_mat*N_mat;
	hipMemcpy(c, dev_c, mem_size_C, hipMemcpyDeviceToHost);
	printmatrix(c);
	printmatrix(a);
	printmatrix(b);


	//Retrieve timer
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Time (Fast): %f ms \n", milliseconds);
	std::cout << std::endl;


	//Slow Serial Execution
	if (!DEBUG) {
		block_size = 1;
		hipEventRecord(start1);
		matrixMulCUDASlow << <iterations, 1 >> > (dev_a, dev_b, dev_c_slow);
		hipEventRecord(stop1);

		//Occupancy
		int output = 1;
		hipOccupancyMaxActiveBlocksPerMultiprocessor(
			&output,
			matrixMulCUDASlow,
			block_size,
			0);

		double activeWarps = (double)output * (double)block_size / (double)prop.warpSize;
		double maxWarps = (double)prop.maxThreadsPerMultiProcessor / (double)prop.warpSize;

		std::cout << "Occupancy (slow): " << (double)activeWarps / maxWarps * 100 << "%" << std::endl;
		std::cout << std::endl;


		//Slow Print
		std::cout << "Slow Serial Execution:\n";
		unsigned long mem_size_C = sizeof(int) *N_mat*N_mat;
		hipMemcpy(cslow, dev_c_slow, mem_size_C, hipMemcpyDeviceToHost);
		printmatrix(cslow);

		//Retrieve slowtimer
		hipEventSynchronize(stop1);
		float millisecondsslow = 0;
		hipEventElapsedTime(&millisecondsslow, start1, stop1);
		printf("Time (Slow): %f ms \n", millisecondsslow);

		std::cout << std::endl;

		printf("Speedup: %f times\n", (millisecondsslow / milliseconds));
	}

	//Free memory
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_c_slow);

	hipError_t cudaStatus;
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}


	std::cout << "Press ENTER to exit...";
	std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
	return 0;
}

