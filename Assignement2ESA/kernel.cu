#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


//template <int BLOCK_SIZE> __global__ void




uint32_t h_C[169] = { 0 };


__global__ void matrixMulCUDA(int *A, int *B, int *C)
{
	//const int BLOCK_SIZE = 13;
	// Block index
	//int bx = blockIdx.x;
	//int by = blockIdx.y;

	// Thread index
	int row = threadIdx.x;
	int col = threadIdx.y;

	int multi = 0;

	for (int j = 0; j < 13; j++) {
		multi += A[(row * 13) + j] * B[col + (13 * j)];
	}
	__syncthreads();
	C[(row*13)+col] = multi + A[(row * 13) + col] + B[(row * 13)+col];
}


int main()
{
	int a[169] = {
		28, 122,  80,   42,   54,   122,  98,   42,   99,   58,   124,  29,   21 ,
		113,  85,   30,   35,   41,   98,   103,  68,   15,   50,   31,   80,   54,
		47, 37,   23,   96,   59,   47,   84,   26,   84,   72,   51,   118,  119 ,
		38, 121,  45,   21,   87,   91,   20,   69,   98,   119,  15,   89,   47 ,
		40, 71,   105,  76,   31,   65,   109,  30,   127,  110,  17,   64,   64 ,
		45, 20,   113,  86,   86,   51,   104,  115,  61,   103,  60,   113,  44 ,
		101,  107,  33,   63,   39,   47,   120,  20,   41,   64,   102,  59,   86 ,
		9,  42,   118,  26,   83,   123,  10,   82,   47,   108,  127,  4,    66 ,
		75, 26,   117,  80,   47,   111,  38,   22,   98,   101,  92,   100,  48 ,
		90, 9,    71,   36,   90,   95,   4,    94,   72,   29,   77,   118,  78 ,
		81, 75,   97,   127,  22,   8,    96,   80,   100,  88,   69,   114,  16 ,
		25, 109,  74,   3,    126,  56,   99,   15,   69,   73,   76,   19,   97 ,
		59, 84,   102,  53,   30,   34,   33,   105,  75,   102,  60,   121,  93
	};

	int b[169] = {
		102,  61,   111,  79,   99,   3,    25,   50,   33,   48,   5,    94,   28 ,
		106,  89,   35,   37,   112   ,51   ,13,  70,   3,    110,  31,   7,    99 ,
		65, 115,  94,   68,   95,   114,  34,   34,   64,   1,    11,   66,   126 ,
		114,  37,   42,   3,    88,   35,   124   ,50,  74,   95,   25,   34    ,24 ,
		25, 111,  4,    116   ,54,  90    ,11   ,32   ,121, 20,   26,   62,   60 ,
		45, 41,   20,   33,   89,   75,   89,   2,    28    ,19,  96,   46,   119 ,
		39, 68,   87,   59,   33,   82    ,94,  14,   115,  0,    0,    92,   85 ,
		58, 62,   122,  106   ,93,  39,   86,   80,   75,   23,   57,   89,   7 ,
		119,  75,   20,   42,   1,    120,  83,   24,   62,   78,   20,   25,   126 ,
		121,  42,   78,   45,   8,    17,   52,   38,   44,   13,   104,  57,   62 ,
		29, 96,   0,    64,   47,   50,   22,   17,   88,   63,   108,  78,   101 ,
		70, 108,  69,   12,   0,    80,   115   ,107, 71,   54,   5,    57,   3 ,
		123,  72,   56,   5,    30,   45    ,2,   11,   124,  84,   63,   47,   104
	};

	int c[169] = { 0 };

	int *dev_a, *dev_b, *dev_c;

	hipMalloc((void**)&dev_a, 169 *sizeof(int));
	hipMalloc((void**)&dev_b, 169 * sizeof(int));
	hipMalloc((void**)&dev_c, 169 * sizeof(int));
	
	hipMemcpy(dev_a, a, 169 *sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, 169 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, 169 * sizeof(int), hipMemcpyHostToDevice);


	int block_size = 13;
	// Setup execution parameters
	dim3 threads(block_size, block_size);
	dim3 grid(13 / threads.x, 13 / threads.y);

	matrixMulCUDA <<<1,threads>>> (dev_a, dev_b, dev_c);

	unsigned long mem_size_C = sizeof(int) * 169;
	hipMemcpy(c, dev_c, mem_size_C, hipMemcpyDeviceToHost);
	for(int i=0; i<169; i++)
	printf("value %d \n", c[i]);


	hipError_t cudaStatus;
	//const int arraySize = 5;
	//const int a[arraySize] = { 1, 2, 3, 4, 5 };
	//const int b[arraySize] = { 10, 20, 30, 40, 50 };
	//int c[arraySize] = { 0 };
	//// Add vectors in parallel.
	//cudaStatus = addWithCuda(c, a, b, arraySize);
	//if (cudaStatus != hipSuccess) {
	//    fprintf(stderr, "addWithCuda failed!");
	//    return 1;
	//}

	//printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
	//    c[0], c[1], c[2], c[3], c[4]);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

